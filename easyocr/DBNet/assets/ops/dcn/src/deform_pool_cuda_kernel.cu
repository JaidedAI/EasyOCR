#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2017 Microsoft
 * Licensed under The MIT License [see LICENSE for details]
 * \file deformable_psroi_pooling.cu
 * \brief
 * \author Yi Li, Guodong Zhang, Jifeng Dai
*/
/***************** Adapted by Charles Shang *********************/
// modify from https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/blob/mmdetection/mmdet/ops/dcn/src/cuda/deform_psroi_pooling_cuda.cu

#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>
#include <stdio.h>
#include <math.h>
#include <algorithm>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                        \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n);                                       \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N)
{
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename scalar_t>
__device__ scalar_t bilinear_interp(
    const scalar_t *data,
    const scalar_t x,
    const scalar_t y,
    const int width,
    const int height)
{
  int x1 = floor(x);
  int x2 = ceil(x);
  int y1 = floor(y);
  int y2 = ceil(y);
  scalar_t dist_x = (scalar_t)(x - x1);
  scalar_t dist_y = (scalar_t)(y - y1);
  scalar_t value11 = data[y1 * width + x1];
  scalar_t value12 = data[y2 * width + x1];
  scalar_t value21 = data[y1 * width + x2];
  scalar_t value22 = data[y2 * width + x2];
  scalar_t value = (1 - dist_x) * (1 - dist_y) * value11 + (1 - dist_x) * dist_y * value12 + dist_x * (1 - dist_y) * value21 + dist_x * dist_y * value22;
  return value;
}

template <typename scalar_t>
__global__ void DeformablePSROIPoolForwardKernel(
    const int count,
    const scalar_t *bottom_data,
    const scalar_t spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const scalar_t *bottom_rois, const scalar_t *bottom_trans,
    const int no_trans,
    const scalar_t trans_std,
    const int sample_per_part,
    const int output_dim,
    const int group_size,
    const int part_size,
    const int num_classes,
    const int channels_each_class,
    scalar_t *top_data,
    scalar_t *top_count)
{
  CUDA_KERNEL_LOOP(index, count)
  {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const scalar_t *offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    scalar_t roi_start_w = (scalar_t)(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
    scalar_t roi_start_h = (scalar_t)(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
    scalar_t roi_end_w = (scalar_t)(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
    scalar_t roi_end_h = (scalar_t)(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

    // Force too small ROIs to be 1x1
    scalar_t roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
    scalar_t roi_height = max(roi_end_h - roi_start_h, 0.1);

    // Compute w and h at bottom
    scalar_t bin_size_h = roi_height / (scalar_t)(pooled_height);
    scalar_t bin_size_w = roi_width / (scalar_t)(pooled_width);

    scalar_t sub_bin_size_h = bin_size_h / (scalar_t)(sample_per_part);
    scalar_t sub_bin_size_w = bin_size_w / (scalar_t)(sample_per_part);

    int part_h = floor((scalar_t)(ph) / pooled_height * part_size);
    int part_w = floor((scalar_t)(pw) / pooled_width * part_size);
    int class_id = ctop / channels_each_class;
    scalar_t trans_x = no_trans ? (scalar_t)(0) : bottom_trans[(((n * num_classes + class_id) * 2) * part_size + part_h) * part_size + part_w] * (scalar_t)trans_std;
    scalar_t trans_y = no_trans ? (scalar_t)(0) : bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w] * (scalar_t)trans_std;

    scalar_t wstart = (scalar_t)(pw)*bin_size_w + roi_start_w;
    wstart += trans_x * roi_width;
    scalar_t hstart = (scalar_t)(ph)*bin_size_h + roi_start_h;
    hstart += trans_y * roi_height;

    scalar_t sum = 0;
    int count = 0;
    int gw = floor((scalar_t)(pw)*group_size / pooled_width);
    int gh = floor((scalar_t)(ph)*group_size / pooled_height);
    gw = min(max(gw, 0), group_size - 1);
    gh = min(max(gh, 0), group_size - 1);

    const scalar_t *offset_bottom_data = bottom_data + (roi_batch_ind * channels) * height * width;
    for (int ih = 0; ih < sample_per_part; ih++)
    {
      for (int iw = 0; iw < sample_per_part; iw++)
      {
        scalar_t w = wstart + iw * sub_bin_size_w;
        scalar_t h = hstart + ih * sub_bin_size_h;
        // bilinear interpolation
        if (w < -0.5 || w > width - 0.5 || h < -0.5 || h > height - 0.5)
        {
          continue;
        }
        w = min(max(w, 0.), width - 1.);
        h = min(max(h, 0.), height - 1.);
        int c = (ctop * group_size + gh) * group_size + gw;
        scalar_t val = bilinear_interp(offset_bottom_data + c * height * width, w, h, width, height);
        sum += val;
        count++;
      }
    }
    top_data[index] = count == 0 ? (scalar_t)(0) : sum / count;
    top_count[index] = count;
  }
}

template <typename scalar_t>
__global__ void DeformablePSROIPoolBackwardAccKernel(
    const int count,
    const scalar_t *top_diff,
    const scalar_t *top_count,
    const int num_rois,
    const scalar_t spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int output_dim,
    scalar_t *bottom_data_diff, scalar_t *bottom_trans_diff,
    const scalar_t *bottom_data,
    const scalar_t *bottom_rois,
    const scalar_t *bottom_trans,
    const int no_trans,
    const scalar_t trans_std,
    const int sample_per_part,
    const int group_size,
    const int part_size,
    const int num_classes,
    const int channels_each_class)
{
  CUDA_KERNEL_LOOP(index, count)
  {
    // The output is in order (n, ctop, ph, pw)
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int ctop = (index / pooled_width / pooled_height) % output_dim;
    int n = index / pooled_width / pooled_height / output_dim;

    // [start, end) interval for spatial sampling
    const scalar_t *offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];
    scalar_t roi_start_w = (scalar_t)(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
    scalar_t roi_start_h = (scalar_t)(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
    scalar_t roi_end_w = (scalar_t)(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
    scalar_t roi_end_h = (scalar_t)(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

    // Force too small ROIs to be 1x1
    scalar_t roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
    scalar_t roi_height = max(roi_end_h - roi_start_h, 0.1);

    // Compute w and h at bottom
    scalar_t bin_size_h = roi_height / (scalar_t)(pooled_height);
    scalar_t bin_size_w = roi_width / (scalar_t)(pooled_width);

    scalar_t sub_bin_size_h = bin_size_h / (scalar_t)(sample_per_part);
    scalar_t sub_bin_size_w = bin_size_w / (scalar_t)(sample_per_part);

    int part_h = floor((scalar_t)(ph) / pooled_height * part_size);
    int part_w = floor((scalar_t)(pw) / pooled_width * part_size);
    int class_id = ctop / channels_each_class;
    scalar_t trans_x = no_trans ? (scalar_t)(0) : bottom_trans[(((n * num_classes + class_id) * 2) * part_size + part_h) * part_size + part_w] * (scalar_t)trans_std;
    scalar_t trans_y = no_trans ? (scalar_t)(0) : bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w] * (scalar_t)trans_std;

    scalar_t wstart = (scalar_t)(pw)*bin_size_w + roi_start_w;
    wstart += trans_x * roi_width;
    scalar_t hstart = (scalar_t)(ph)*bin_size_h + roi_start_h;
    hstart += trans_y * roi_height;

    if (top_count[index] <= 0)
    {
      continue;
    }
    scalar_t diff_val = top_diff[index] / top_count[index];
    const scalar_t *offset_bottom_data = bottom_data + roi_batch_ind * channels * height * width;
    scalar_t *offset_bottom_data_diff = bottom_data_diff + roi_batch_ind * channels * height * width;
    int gw = floor((scalar_t)(pw)*group_size / pooled_width);
    int gh = floor((scalar_t)(ph)*group_size / pooled_height);
    gw = min(max(gw, 0), group_size - 1);
    gh = min(max(gh, 0), group_size - 1);

    for (int ih = 0; ih < sample_per_part; ih++)
    {
      for (int iw = 0; iw < sample_per_part; iw++)
      {
        scalar_t w = wstart + iw * sub_bin_size_w;
        scalar_t h = hstart + ih * sub_bin_size_h;
        // bilinear interpolation
        if (w < -0.5 || w > width - 0.5 || h < -0.5 || h > height - 0.5)
        {
          continue;
        }
        w = min(max(w, 0.), width - 1.);
        h = min(max(h, 0.), height - 1.);
        int c = (ctop * group_size + gh) * group_size + gw;
        // backward on feature
        int x0 = floor(w);
        int x1 = ceil(w);
        int y0 = floor(h);
        int y1 = ceil(h);
        scalar_t dist_x = w - x0, dist_y = h - y0;
        scalar_t q00 = (1 - dist_x) * (1 - dist_y);
        scalar_t q01 = (1 - dist_x) * dist_y;
        scalar_t q10 = dist_x * (1 - dist_y);
        scalar_t q11 = dist_x * dist_y;
        int bottom_index_base = c * height * width;
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y0 * width + x0, q00 * diff_val);
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y1 * width + x0, q01 * diff_val);
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y0 * width + x1, q10 * diff_val);
        atomicAdd(offset_bottom_data_diff + bottom_index_base + y1 * width + x1, q11 * diff_val);

        if (no_trans)
        {
          continue;
        }
        scalar_t U00 = offset_bottom_data[bottom_index_base + y0 * width + x0];
        scalar_t U01 = offset_bottom_data[bottom_index_base + y1 * width + x0];
        scalar_t U10 = offset_bottom_data[bottom_index_base + y0 * width + x1];
        scalar_t U11 = offset_bottom_data[bottom_index_base + y1 * width + x1];
        scalar_t diff_x = (U11 * dist_y + U10 * (1 - dist_y) - U01 * dist_y - U00 * (1 - dist_y)) * trans_std * diff_val;
        diff_x *= roi_width;
        scalar_t diff_y = (U11 * dist_x + U01 * (1 - dist_x) - U10 * dist_x - U00 * (1 - dist_x)) * trans_std * diff_val;
        diff_y *= roi_height;

        atomicAdd(bottom_trans_diff + (((n * num_classes + class_id) * 2) * part_size + part_h) * part_size + part_w, diff_x);
        atomicAdd(bottom_trans_diff + (((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w, diff_y);
      }
    }
  }
}

void DeformablePSROIPoolForward(const at::Tensor data,
                                const at::Tensor bbox,
                                const at::Tensor trans,
                                at::Tensor out,
                                at::Tensor top_count,
                                const int batch,
                                const int channels,
                                const int height,
                                const int width,
                                const int num_bbox,
                                const int channels_trans,
                                const int no_trans,
                                const float spatial_scale,
                                const int output_dim,
                                const int group_size,
                                const int pooled_size,
                                const int part_size,
                                const int sample_per_part,
                                const float trans_std)
{
  const int pooled_height = pooled_size;
  const int pooled_width = pooled_size;
  const int count = num_bbox * output_dim * pooled_height * pooled_width;
  const int num_classes = no_trans ? 1 : channels_trans / 2;
  const int channels_each_class = no_trans ? output_dim : output_dim / num_classes;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data.type(), "deformable_psroi_pool_forward", ([&] {
        const scalar_t *bottom_data = data.data<scalar_t>();
        const scalar_t *bottom_rois = bbox.data<scalar_t>();
        const scalar_t *bottom_trans = no_trans ? NULL : trans.data<scalar_t>();
        scalar_t *top_data = out.data<scalar_t>();
        scalar_t *top_count_data = top_count.data<scalar_t>();

        DeformablePSROIPoolForwardKernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
            count, bottom_data, (scalar_t)spatial_scale, channels, height, width, pooled_height, pooled_width,
            bottom_rois, bottom_trans, no_trans, (scalar_t)trans_std, sample_per_part, output_dim,
            group_size, part_size, num_classes, channels_each_class, top_data, top_count_data);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in DeformablePSROIPoolForward: %s\n", hipGetErrorString(err));
  }
}

void DeformablePSROIPoolBackwardAcc(const at::Tensor out_grad,
                                    const at::Tensor data,
                                    const at::Tensor bbox,
                                    const at::Tensor trans,
                                    const at::Tensor top_count,
                                    at::Tensor in_grad,
                                    at::Tensor trans_grad,
                                    const int batch,
                                    const int channels,
                                    const int height,
                                    const int width,
                                    const int num_bbox,
                                    const int channels_trans,
                                    const int no_trans,
                                    const float spatial_scale,
                                    const int output_dim,
                                    const int group_size,
                                    const int pooled_size,
                                    const int part_size,
                                    const int sample_per_part,
                                    const float trans_std)
{
  // LOG(INFO) << "DeformablePSROIPoolBackward";
  const int num_rois = num_bbox;
  const int pooled_height = pooled_size;
  const int pooled_width = pooled_size;
  const int count = num_bbox * output_dim * pooled_height * pooled_width;
  const int num_classes = no_trans ? 1 : channels_trans / 2;
  const int channels_each_class = no_trans ? output_dim : output_dim / num_classes;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      out_grad.type(), "deformable_psroi_pool_backward_acc", ([&] {
        const scalar_t *top_diff = out_grad.data<scalar_t>();
        const scalar_t *bottom_data = data.data<scalar_t>();
        const scalar_t *bottom_rois = bbox.data<scalar_t>();
        const scalar_t *bottom_trans = no_trans ? NULL : trans.data<scalar_t>();
        scalar_t *bottom_data_diff = in_grad.data<scalar_t>();
        scalar_t *bottom_trans_diff = no_trans ? NULL : trans_grad.data<scalar_t>();
        const scalar_t *top_count_data = top_count.data<scalar_t>();

        DeformablePSROIPoolBackwardAccKernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(
            count, top_diff, top_count_data, num_rois, (scalar_t)spatial_scale, channels, height, width,
            pooled_height, pooled_width, output_dim, bottom_data_diff, bottom_trans_diff,
            bottom_data, bottom_rois, bottom_trans, no_trans, (scalar_t)trans_std, sample_per_part,
            group_size, part_size, num_classes, channels_each_class);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("error in DeformablePSROIPoolForward: %s\n", hipGetErrorString(err));
  }
}